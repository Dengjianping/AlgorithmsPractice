#include "hip/hip_runtime.h"
int row = blockDim.y * blockIdx.y + threadIdx.y;
int col = blockDim.x * blockIdx.x + threadIdx.x;

extern __shared__ int shared[TILE_H + K_SIZE - 1][TILE_W + K_SIZE - 1];
if (row < height && col < width)
{
    if (row - radius < 0 || col - radius < 0)
        shared[threadIdx.y][threadIdx.x] = 0;
    else
        shared[threadIdx.y][threadIdx.x] = *((int*)((char*)input + (row - radius) * pitch) + (col - radius));
    
    if (row - radius < 0 || col + radius > width - 1)
        shared[threadIdx.y][threadIdx.x + blockDim.x] = 0;
    else
        shared[threadIdx.y][threadIdx.x + blockDim.x] = *((int*)((char*)input + (row - radius) * pitch) + (col + radius));
    
    if (row + radius > height -1 || col - radius < 0)
        shared[thread.y + blockDim.y][threadIdx.x] = 0;
    else
        shared[thread.y + blockDim.y][threadIdx.x] = *((int*)((char*)input + (row + radius) * pitch) + (col - radius));
    
    if (row + radius > height - 1 || col - radius > width - 1)
        shared[threadIdx.y + blockDimy][threadIdx.x + blockDim.x] = 0;
    else
        shared[threadIdx.y + blockDimy][threadIdx.x + blockDim.x] = *((int*)((char*)input + (row + radius) * pitch) + (col + radius));
    __syncthreads();
    
    int sum = 0;
    for (int i = -radius; i <= radius; i++)
        for (int j = -radius; j <= radius; j++)
        {
            int s1= shared[threadIdx.y + radius - i][threadIdx.x + radius - j] * kernel[radius + i][radius + j]; // ILP optimation
            int s2= shared[threadIdx.y + radius - i][threadIdx.x + radius - j] * kernel[radius + i][radius + j]; // ILP optimation
            sum += s1 + s2;
        }
        
    int *out = (int*)((char)input + row * pitch) + col;
    *out = sum;
}